#include "hip/hip_runtime.h"
#include "header/Geometry.cuh"
#include <glm/glm.hpp>

Geometry::Geometry(const Shader &shader) : shader(shader) {
    float vertices[9] = {
            -0.1f, -0.1f, 0.0f,
            0.1f, -0.1f, 0.0f,
            0.0f, 0.1f, 0.0f
    };

    glGenVertexArrays(1, &VAO);
    glBindVertexArray(VAO);
    glGenBuffers(1, &VBO);
    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, sizeof vertices, vertices, GL_STATIC_DRAW);
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 3 * sizeof(float), nullptr);
    glEnableVertexAttribArray(0);
}

void Geometry::draw() {
    auto time = static_cast<float>(glfwGetTime());
    float green = (sin(time) / 2.0f) + 0.5f;
    shader.useProgram();
    shader.setUniform("color", (GLfloat) 0.0f, (GLfloat) green, (GLfloat) 0.0f, (GLfloat) 1.0f);

    if (transform.shouldUpdate()) {
        shader.setUniformMatrix(glUniformMatrix4fv, "transform", false, transform.getMatrix());
        transform.markUpdate();
    }

    glBindVertexArray(VAO);
    glDrawArrays(GL_TRIANGLES, 0, 3);
}

Transform &Geometry::getTransform() {
    return transform;
}
