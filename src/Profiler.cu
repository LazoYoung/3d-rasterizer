#include "hip/hip_runtime.h"
#include "Profiler.cuh"

int Profiler::getFramesPerSecond() const {
    return _fps;
}

void Profiler::updateFrameRate() {
    auto thisFrame = high_resolution_clock::now();
    auto diff = duration_cast<milliseconds>(thisFrame - _lastFrame).count();
    _timer += diff;
    _lastFrame = thisFrame;

    if (_timer > _delay) {
        _fps = diff > 0 ? static_cast<int>(floor(1000.0f / diff)) : 0;
        _timer = 0;
    }
}

void Profiler::startChrono() {
    _state = RUN;
    _startPoint = high_resolution_clock::now();
}

void Profiler::stopChrono() {
    _state = STOP;
    _stopPoint = high_resolution_clock::now();
}

long long Profiler::getElapsed() {
    if (_state == RUN) {
        stopChrono();
    }

    return duration_cast<milliseconds>(_stopPoint - _startPoint).count();
}
