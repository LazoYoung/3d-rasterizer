#include "hip/hip_runtime.h"
#include "Profiler.cuh"

int Profiler::getFramesPerSecond() const {
    return _fps;
}

void Profiler::updateFrameRate() {
    auto now = high_resolution_clock::now();
    auto diff = duration_cast<milliseconds>(now - _lastTime).count();
    _timer += diff;
    _lastTime = now;

    if (_timer > _delay) {
        _fps = diff > 0 ? static_cast<int>(floor(1000.0f / diff)) : 0;
        _timer = 0;
    }
}
