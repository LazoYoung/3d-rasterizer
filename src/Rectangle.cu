#include "hip/hip_runtime.h"
#include "header/Rectangle.cuh"

Rectangle::Rectangle() : Geometry(_array, sizeof _array) {}

vec4 Rectangle::getColor() {
    auto time = static_cast<float>(glfwGetTime());
    float green = (sin(time) / 2.0f) + 0.5f;
    return {0.0f, green, 0.0f, 1.0f};
}

void Rectangle::bind() {
    Geometry::bind();

    glGenBuffers(1, &EBO);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof _indices, _indices, GL_STATIC_DRAW);
}

void Rectangle::draw() {
    glBindVertexArray(VAO);
    glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, nullptr);
}
