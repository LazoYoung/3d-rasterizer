#include "hip/hip_runtime.h"
#include "header/Triangle.cuh"

Triangle::Triangle() : Geometry(array, sizeof array) {}

vec4 Triangle::getColor() {
    auto time = static_cast<float>(glfwGetTime());
    float green = (sin(time) / 2.0f) + 0.5f;
    return {0.0f, green, 0.0f, 1.0f};
}
