#include "hip/hip_runtime.h"
#include "Geometry.cuh"
#include "glm/glm.hpp"
#include <hip/hip_vector_types.h>
#include <vector_functions.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

Geometry::Geometry(const GLfloat *vertexArray, GLsizeiptr vertexSize, GLsizei vertexCount) :
        vertexArray(vertexArray),
        vertexSize(vertexSize),
        vertexCount(vertexCount) {
    _transform.setUpdateCallback([this] { resetModel(); });
}

void Geometry::bind() {
    if (_isBound) return;

    glGenVertexArrays(1, &VAO);
    glBindVertexArray(VAO);
    glGenBuffers(1, &VBO);
    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    glBufferData(GL_ARRAY_BUFFER, vertexSize, vertexArray, GL_STATIC_DRAW);
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 3 * sizeof(GLfloat), nullptr);
    glEnableVertexAttribArray(0);
    cudaCheckError(hipMalloc(&cudaVertexArray, vertexSize));
    cudaCheckError(hipMemcpy(cudaVertexArray, vertexArray, vertexSize, hipMemcpyHostToDevice));
}

vec4 Geometry::getColor() {
    return {1.0f, 1.0f, 1.0f, 1.0f};
}

void Geometry::render(Scene *scene) {
    bind();
    updateShader(scene);
    processVertex(scene);
    draw();
}

void Geometry::updateShader(Scene *scene) {
    Shader *shader = scene->getShader();
    Camera &camera = scene->getCamera();
    vec4 color = getColor();
    bool decouple = shader->getPipeline() != OpenGL;

    if (!decouple) {
        mat4 &model = getModel();
        mat4 &view = camera.getView();
        mat4 &projection = camera.getProjection();
        shader->setUniformMatrix(glUniformMatrix4fv, "model", false, model);
        shader->setUniformMatrix(glUniformMatrix4fv, "view", false, view);
        shader->setUniformMatrix(glUniformMatrix4fv, "projection", false, projection);
    }

    shader->setUniform("decouple", decouple);
    shader->setUniform("color", color.x, color.y, color.z, color.w);
}

void Geometry::processVertex(Scene *scene) {
    Pipeline pipeline = scene->getShader()->getPipeline();

    switch (pipeline) {
        case CUDA:
            processVertexCuda(scene);
            break;
        case OpenMP:
            processVertexOpenMP(scene);
            break;
    }
}

Transform &Geometry::getTransform() {
    return _transform;
}

mat4 &Geometry::getModel() {
    if (_model) {
        return *_model;
    }

    static auto unitX = vec3(1.0f, 0.0f, 0.0f);
    static auto unitY = vec3(0.0f, 1.0f, 0.0f);
    static auto unitZ = vec3(0.0f, 0.0f, 1.0f);
    const vec3 &rotation = _transform.getRotation();
    _model = new mat4(1.0f);
    mat4 &model = *_model;
    model = glm::translate(model, _transform.getPosition());
    model = glm::rotate(model, radians(rotation.x), unitX);
    model = glm::rotate(model, radians(rotation.y), unitY);
    model = glm::rotate(model, radians(rotation.z), unitZ);
    model = glm::scale(model, _transform.getScale());
    return model;
}

void Geometry::resetModel() {
    if (_model) {
        free(_model);
        _model = nullptr;
    }
}

__global__ void transformFromKernel(const float *d_vertices, const float *d_transform, float *d_result, int vertexCount) {
    unsigned int vertexIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if (vertexIndex < vertexCount) {
        unsigned int idx = vertexIndex * 3;
        float4 pos = make_float4(d_vertices[idx], d_vertices[idx + 1], d_vertices[idx + 2], 1.0f);
        float4 result;

        result.x = d_transform[0] * pos.x + d_transform[4] * pos.y + d_transform[8] * pos.z + d_transform[12] * pos.w;
        result.y = d_transform[1] * pos.x + d_transform[5] * pos.y + d_transform[9] * pos.z + d_transform[13] * pos.w;
        result.z = d_transform[2] * pos.x + d_transform[6] * pos.y + d_transform[10] * pos.z + d_transform[14] * pos.w;
        result.w = d_transform[3] * pos.x + d_transform[7] * pos.y + d_transform[11] * pos.z + d_transform[15] * pos.w;

//        result.x = d_transform[0] * pos.x + d_transform[1] * pos.y + d_transform[2] * pos.z + d_transform[3] * pos.w;
//        result.y = d_transform[4] * pos.x + d_transform[5] * pos.y + d_transform[6] * pos.z + d_transform[7] * pos.w;
//        result.z = d_transform[8] * pos.x + d_transform[9] * pos.y + d_transform[10] * pos.z + d_transform[11] * pos.w;
//        result.w = d_transform[12] * pos.x + d_transform[13] * pos.y + d_transform[14] * pos.z + d_transform[15] * pos.w;

        d_result[idx] = result.x;
        d_result[idx + 1] = result.y;
        d_result[idx + 2] = result.z;
    }
}

void Geometry::processVertexOpenMP(Scene *scene) {
    Camera &camera = scene->getCamera();
    mat4 &model = getModel();
    mat4 &view = camera.getView();
    mat4 &proj = camera.getProjection();
    mat4 transform = proj * view * model;

    glBindBuffer(GL_ARRAY_BUFFER, VBO);
    auto *vertices = static_cast<GLfloat *>(glMapBuffer(GL_ARRAY_BUFFER, GL_READ_ONLY));

    for (int i = 0; i < vertexCount; ++i) {
        vec4 vertex(vertices[i * 3], vertices[i * 3 + 1], vertices[i * 3 + 2], 1.0f);
        scene->getShader()->useProgram();
        scene->getShader()->setUniformVector(glUniform3fv, "dPosition", vertex);
    }

    glUnmapBuffer(GL_ARRAY_BUFFER);
    glBindBuffer(GL_ARRAY_BUFFER, 0);
}

void Geometry::processVertexCuda(Scene *scene) {
    hipGraphicsResource *cudaVBO;
    hipGraphicsGLRegisterBuffer(&cudaVBO, VBO, hipGraphicsRegisterFlagsNone);
    hipGraphicsMapResources(1, &cudaVBO, nullptr);

    // todo: restore VBO if CUDA pipeline is disabled
    Camera &camera = scene->getCamera();
    mat4 &model = getModel();
    mat4 &view = camera.getView();
    mat4 &proj = camera.getProjection();
//    mat4 h_transform = proj * view * model;
    mat4 h_transform = glm::transpose(proj * view * model);

    // device memory allocation
    float *d_transform, *d_vertices;
    hipGraphicsResourceGetMappedPointer(reinterpret_cast<void **>(&d_vertices), nullptr, cudaVBO);
    cudaCheckError(hipMalloc(&d_transform, sizeof(float) * 16));

    // host to device
    cudaCheckError(hipMemcpy(d_transform, glm::value_ptr(h_transform), 16 * sizeof(float), hipMemcpyHostToDevice));

    dim3 blockSize(256);
    dim3 gridSize((vertexCount + blockSize.x - 1) / blockSize.x);
    transformFromKernel<<<gridSize, blockSize>>>(cudaVertexArray, d_transform, d_vertices, vertexCount);
    hipDeviceSynchronize();

    // release memory
    cudaCheckError(hipFree(d_transform));
    hipGraphicsUnregisterResource(cudaVBO);
}

void Geometry::cudaCheckError(hipError_t err) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}
