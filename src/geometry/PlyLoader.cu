#include "hip/hip_runtime.h"
#include "PlyLoader.cuh"
#include "glm/glm.hpp"
#include <fstream>
#include <sstream>
#include <stdexcept>
#include <string>
#include <iostream>
#include <algorithm>

Model PlyLoader::importModel(const char *filePath) {
    auto *vertexSet = new ModelVertex();
    auto *faceSet = new ModelFace();

    readPlyFile(filePath, *vertexSet, *faceSet);
    return {vertexSet, faceSet};
}

void PlyLoader::readPlyFile(const char *path, ModelVertex &vert, ModelFace &face) {
    ifstream file(path);

    if (!file.is_open()) {
        throw invalid_argument(string("Failed to open file: ").append(path));
    }

    cout << "Loading model: " << path << '\n';

    bool bakeNorms;
    processHeader(file, vert, face, bakeNorms);
    processVertex(file, vert, bakeNorms);
    processFace(file, face);

    if (bakeNorms) {
        bakeNormals(vert, face);
    }

    cout << "Found " << vert.count << " vertex points and " << face.count << " meshes." << endl;
}

void PlyLoader::processVertex(ifstream &file, ModelVertex &vert, bool bakeNormals) const {
    string line;
    int vertexIdx = 0;
    int keyCount = static_cast<int>(vert.keys.size());
    vert.arrayCount = vert.count * keyCount;
    vert.arraySize = vert.arrayCount * sizeof(GLfloat);
    vert.vertices = new GLfloat [vert.arrayCount];
    memset(vert.vertices, 0, vert.arraySize);

    while (getline(file, line)) {
        istringstream stream(line);

        for (int i = 0; i < keyCount; ++i) {
            if (bakeNormals && vert.keyIndex["nx"] <= i) {
                break;
            }

            GLfloat vertex;
            stream >> vertex;
            vert.vertices[vertexIdx * keyCount + i] = vertex;

            if (_verbose) {
                cout << vertex << ' ';
            }
        }

        if (_verbose) {
            cout << endl;
        }

        if (++vertexIdx >= vert.count) {
            break;
        }
    }
}

void PlyLoader::bakeNormals(ModelVertex &vert, const ModelFace &face) {
    switch (_device) {
        case CPU:
            bakeNormalsFromCPU(vert, face);
            break;
        case CUDA:
            bakeNormalsFromCUDA(vert, face);
            break;
    }
}

void PlyLoader::bakeNormalsFromCPU(ModelVertex &v, const ModelFace &f) {
    _profiler.startChrono();

    vector<vec3> normals(v.count);
    auto stride = v.keys.size();

    #pragma omp parallel for
    for (int tID = 0; tID < f.count; ++tID) {
        auto v1 = f.indices[tID * 3];
        auto v2 = f.indices[tID * 3 + 1];
        auto v3 = f.indices[tID * 3 + 2];
        vec3 A = vec3(v.vertices[v1 * stride], v.vertices[v1 * stride + 1], v.vertices[v1 * stride + 2]);
        vec3 B = vec3(v.vertices[v2 * stride], v.vertices[v2 * stride + 1], v.vertices[v2 * stride + 2]);
        vec3 C = vec3(v.vertices[v3 * stride], v.vertices[v3 * stride + 1], v.vertices[v3 * stride + 2]);

        // Cross product on 2 vectors that form a polygon
        vec3 normal = glm::cross(B - A, C - A);

        // Accumulate normals per vertex
        #pragma omp critical
        {
            normals[v1] += normal;
            normals[v2] += normal;
            normals[v3] += normal;
        }
    }

    for (int i = 0; i < v.count; ++i) {
        // Every normal is a unit vector
        vec3 normal = glm::normalize(normals[i]);

        // Assign to vertex array (presuming `nx` key maps to index 3)
        v.vertices[i * stride + 3] = normal.x;
        v.vertices[i * stride + 4] = normal.y;
        v.vertices[i * stride + 5] = normal.z;
    }

    cout << "CPU took " << _profiler.getElapsed() << " milliseconds to bake mesh normals." << endl;
}

__global__ void computeNormalsKernel(float *vertices, const unsigned int *indices, vec3 *normals, int faceCount, int stride) {
    auto tID = blockIdx.x * blockDim.x + threadIdx.x;

    if (tID >= faceCount) return;

    auto v1 = indices[tID * 3];
    auto v2 = indices[tID * 3 + 1];
    auto v3 = indices[tID * 3 + 2];
    vec3 A = vec3(vertices[v1 * stride], vertices[v1 * stride + 1], vertices[v1 * stride + 2]);
    vec3 B = vec3(vertices[v2 * stride], vertices[v2 * stride + 1], vertices[v2 * stride + 2]);
    vec3 C = vec3(vertices[v3 * stride], vertices[v3 * stride + 1], vertices[v3 * stride + 2]);

    // Cross product on 2 vectors that form a polygon
    vec3 normal = cross(B - A, C - A);

    // Accumulate normal of vertex #1
    atomicAdd(&normals[v1].x, normal.x);
    atomicAdd(&normals[v1].y, normal.y);
    atomicAdd(&normals[v1].z, normal.z);

    // Accumulate normal of vertex #2
    atomicAdd(&normals[v2].x, normal.x);
    atomicAdd(&normals[v2].y, normal.y);
    atomicAdd(&normals[v2].z, normal.z);

    // Accumulate normal of vertex #3
    atomicAdd(&normals[v3].x, normal.x);
    atomicAdd(&normals[v3].y, normal.y);
    atomicAdd(&normals[v3].z, normal.z);
}

__global__ void normalizeNormalsKernel(float* vertices, const vec3* normals, int vertexCount, int stride) {
    auto tID = blockIdx.x * blockDim.x + threadIdx.x;

    if (tID >= vertexCount) return;

    // Every normal is a unit vector
    vec3 normal = glm::normalize(normals[tID]);

    // Assign to vertex array (presuming `nx` key maps to index 3)
    vertices[tID * stride + 3] = normal.x;
    vertices[tID * stride + 4] = normal.y;
    vertices[tID * stride + 5] = normal.z;
}

void PlyLoader::bakeNormalsFromCUDA(ModelVertex &v, const ModelFace &f) {
    int vertexCount = v.count;
    int faceCount = f.count;
    int stride = static_cast<int>(v.keys.size());

    // Allocate device memory
    float* d_vertices;
    unsigned int* d_indices;
    vec3* d_normals;
    hipMalloc(&d_vertices, vertexCount * stride * sizeof(float));
    hipMalloc(&d_indices, faceCount * 3 * sizeof(unsigned int));
    hipMalloc(&d_normals, vertexCount * sizeof(vec3));
    hipMemcpy(d_vertices, v.vertices, vertexCount * stride * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_indices, f.indices, faceCount * 3 * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemset(d_normals, 0, vertexCount * sizeof(vec3));

    int blockSize = 256;
    int numBlocksFace = (faceCount + blockSize - 1) / blockSize;
    int numBlocksVertex = (vertexCount + blockSize - 1) / blockSize;

    _profiler.startChrono();

    computeNormalsKernel<<<numBlocksFace, blockSize>>>(d_vertices, d_indices, d_normals, faceCount, stride);
    hipDeviceSynchronize();

    normalizeNormalsKernel<<<numBlocksVertex, blockSize>>>(d_vertices, d_normals, vertexCount, stride);
    hipDeviceSynchronize();

    cout << "CUDA took " << _profiler.getElapsed() << " milliseconds to bake mesh normals." << endl;

    // Copy result to host memory
    hipMemcpy(v.vertices, d_vertices, vertexCount * stride * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_vertices);
    hipFree(d_indices);
    hipFree(d_normals);
}

void PlyLoader::processFace(ifstream &file, ModelFace &face) const {
    string line;
    int faceIdx = 0;
    face.vertexPerFace = getVertexPerFace(file);
    face.arrayCount = face.count * face.vertexPerFace;
    face.arraySize = face.arrayCount * sizeof(GLfloat);
    face.indices = new GLuint [face.arrayCount];
    bool firstLine = true;

    while (getline(file, line)) {
        string firstToken;
        istringstream stream(line);

        if (!firstLine) {
            stream >> firstToken;

            if (stoi(firstToken) != face.vertexPerFace) {
                throw runtime_error("Inconsistent vertex indices!");
            }
        }

        firstLine = false;

        if (_verbose) {
            cout << firstToken << ' ';
        }

        for (int i = 0; i < face.vertexPerFace; ++i) {
            int vertexIndex;
            stream >> vertexIndex;
            face.indices[faceIdx * face.vertexPerFace + i] = vertexIndex;

            if (_verbose) {
                cout << vertexIndex << ' ';
            }
        }

        if (_verbose) {
            cout << '\n';
        }

        if (++faceIdx >= face.count) {
            break;
        }
    }
}

void PlyLoader::processHeader(ifstream &file, ModelVertex &vert, ModelFace &face, bool &bakeNormals) const {
    string line;
    string element;
    int keyIdx = 0;

    while (getline(file, line)) {
        if (line == "end_header") {
            break;
        }

        istringstream lineStream(line);
        string label;

        getline(lineStream, label, ' ');

        if (label == "ply" || label == "format") {
            continue;
        }

        if (label == "element") {
            string count;
            getline(lineStream, element, ' ');
            getline(lineStream, count, ' ');

            if (element == "vertex") {
                vert.count = stoi(count);
            } else if (element == "face") {
                face.count = stoi(count);
            } else {
                throw runtime_error(string("Bad element: ").append(element));
            }

            continue;
        }

        if (label == "property") {
            if (element == "vertex") {
                string dType;
                string dLabel;

                getline(lineStream, dType, ' ');
                getline(lineStream, dLabel, ' ');

                vert.keys.push_back(dLabel);
                vert.keyIndex.insert(pair(dLabel, keyIdx++));
            }

            continue;
        }
    }

    auto &keys = vert.keys;
    bool normalFound = std::find(keys.begin(), keys.end(), "nx") != keys.end();
    bakeNormals = _bakeNorms && !normalFound;
    vert.hasNormals = bakeNormals || normalFound;

    if (bakeNormals) {
        keys.emplace_back("nx");
        vert.keyIndex.insert(pair("nx", keyIdx++));
        keys.emplace_back("ny");
        vert.keyIndex.insert(pair("ny", keyIdx++));
        keys.emplace_back("nz");
        vert.keyIndex.insert(pair("nz", keyIdx++));
    }
}

int PlyLoader::getVertexPerFace(ifstream &file) {
    string str;
    file >> str;
    return stoi(str);
}
