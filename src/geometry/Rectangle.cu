#include "hip/hip_runtime.h"
#include "Rectangle.cuh"

Rectangle::Rectangle() : Geometry(_array, sizeof _array, _count) {}

vec3 Rectangle::getColor() {
    auto time = static_cast<float>(glfwGetTime());
    float green = (sin(time) / 2.0f) + 0.5f;
    return {0.0f, green, 0.0f};
}

void Rectangle::bind(Pipeline pipeline) {
    Geometry::bind(pipeline);

    glGenBuffers(1, &EBO);
    glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, EBO);
    glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof _indices, _indices, GL_STATIC_DRAW);
}

void Rectangle::draw() {
    glBindVertexArray(VAO);
    glDrawElements(GL_TRIANGLES, _count, GL_UNSIGNED_INT, nullptr);
}
