#include "hip/hip_runtime.h"
#include "Camera.cuh"

Camera::Camera() : Camera(45.0f, 800.0f / 600.0f, 0.1f, 100.0f, 2.5f) {}

Camera::Camera(float fov, float aspectRatio, float zNear, float zFar, float speed) :
        _fov(fov),
        _aspect(aspectRatio),
        _zNear(zNear),
        _zFar(zFar),
        _speed(speed) {
    _transform.setPosition(0.0f, 0.0f, 3.0f);
    _transform.setRotation(0.0f, 0.0f, -1.0f);  // forward direction
    _transform.setUpdateCallback([this] { onTransform(); });
}

float Camera::getSpeed() const {
    return _speed;
}

const vec3 &Camera::getUpAxis() {
    return _up;
}

const vec3 &Camera::getRightAxis() {
    if (_right) {
        return *_right;
    }

    vec3 direction = _transform.getRotation();
    _right = new vec3();
    vec3 &right = *_right;
    right = glm::normalize(glm::cross(_up, direction));
    return right;
}

vec3 Camera::getForwardAxis() {
    return glm::normalize(_transform.getRotation());
}

mat4 &Camera::getProjection() {
    if (_projection) {
        return *_projection;
    }

    _projection = new mat4();
    mat4 &proj = *_projection;
    proj = glm::perspective(radians(_fov), _aspect, _zNear, _zFar);
    return proj;
}

mat4 &Camera::getView() {
    if (_view) {
        return *_view;
    }

    static auto unitX = vec3(1.0f, 0.0f, 0.0f);
    static auto unitY = vec3(0.0f, 1.0f, 0.0f);
    static auto unitZ = vec3(0.0f, 0.0f, 1.0f);
    auto &position = _transform.getPosition();
    auto &front = _transform.getRotation();
    _view = new mat4(1.0f);
    mat4 &view = *_view;
    view = glm::lookAt(position, position + front, _up);
    return view;
}

void Camera::move(vec3 velocity) {
    auto &pos = _transform.getPosition();
    float x = pos.x + velocity.x;
    float y = pos.y + velocity.y;
    float z = pos.z + velocity.z;
    _transform.setPosition(x, y, z);
}

void Camera::yaw(double degree) {
    _yaw += degree;

    updateDirection();
}

void Camera::pitch(double degree) {
    _pitch += degree;
    _pitch = clamp(_pitch, -89.0, 89.0);

    updateDirection();
}

void Camera::updateDirection() {
    auto x = static_cast<float>(cos(glm::radians(_yaw)) * cos(glm::radians(_pitch)));
    auto y = static_cast<float>(sin(glm::radians(_pitch)));
    auto z = static_cast<float>(sin(glm::radians(_yaw)) * cos(glm::radians(_pitch)));

    _transform.setRotation(x, y, z);
}

void Camera::onTransform() {
    if (_right) {
        free(_right);
        _right = nullptr;
    }

    if (_view) {
        free(_view);
        _view = nullptr;
    }

    if (_projection) {
        free(_projection);
        _projection = nullptr;
    }
}
